#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

//#define SIZE 1024*1024*1024*20  // Define the size of the vectors
// Error checking macro
#define cudaCheckError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

#define gpuKernelCheck() { gpuKernelAssert(__FILE__, __LINE__); }
inline void gpuKernelAssert(const char* file, int line, bool abort = true) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s %s %d\n", hipGetErrorString(err), file, line);
        if (abort) exit(err);
    }
}

// CUDA Kernel for vector addition
__global__ void vectorAdd(int* A, int* B, int* C, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int* A, * B, * C;            // Host vectors
    int* d_A, * d_B, * d_C;      // Device vectors
    long long SIZE = 1024LL * 1024 * 32;
    long size = SIZE * sizeof(int);

    // CUDA event creation, used for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipError_t err;

    // Allocate device vectors
    cudaCheckError(hipMalloc((void**)&d_A, size));



    err = hipMalloc((void**)&d_B, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
    }



    err = hipMalloc((void**)&d_C, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
    }

    // Allocate and initialize host vectors
    A = (int*)malloc(size);
    B = (int*)malloc(size);
    C = (int*)malloc(size);
    for (int i = 0; i < SIZE; i++) {
        A[i] = i;
        B[i] = SIZE - i;
    }


    // Copy host vectors to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Start recording
    hipEventRecord(start);

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 96;
    int blocksPerGrid = (SIZE + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, SIZE);
    gpuKernelCheck();

    // Stop recording
    hipEventRecord(stop);

    // Copy result back to host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Calculate and print the execution time
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Execution time: %f milliseconds\n", milliseconds);

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}