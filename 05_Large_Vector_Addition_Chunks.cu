#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>

#define TOTAL_SIZE (1024 * 1024 * 1024)  // Total elements
#define CHUNK_SIZE (1024 * 1024 * 128)   // Elements per chunk
#define BLOCK_SIZE 1024                 // Threads per block

// CUDA Kernel
__global__ void vectorAdd(int* a, int* b, int* c, int chunk_size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < chunk_size) {
        c[index] = a[index] + b[index];
    }
}

// Host function to fill array with random ints
void random_ints(int* x, int size) {
    for (int i = 0; i < size; i++) {
        x[i] = rand() % 100;
    }
}

int main() {
    int* d_a, * d_b, * d_c;
    int* chunk_a, * chunk_b, * chunk_c;
    size_t chunkSizeBytes = CHUNK_SIZE * sizeof(int);

    // Allocate device memory
    hipMalloc((void**)&d_a, chunkSizeBytes);
    hipMalloc((void**)&d_b, chunkSizeBytes);
    hipMalloc((void**)&d_c, chunkSizeBytes);
    printf("\nHello 01\n");

    // Allocate host memory
    chunk_a = (int*)malloc(chunkSizeBytes);
    chunk_b = (int*)malloc(chunkSizeBytes);
    chunk_c = (int*)malloc(chunkSizeBytes);

    for (long long offset = 0; offset < TOTAL_SIZE; offset += CHUNK_SIZE) {
        int currentChunkSize = (TOTAL_SIZE - offset) < CHUNK_SIZE ? (TOTAL_SIZE - offset) : CHUNK_SIZE;
        int numBlocks = (currentChunkSize + BLOCK_SIZE - 1) / BLOCK_SIZE;
        printf("\nOffset %lld\n", offset);

        random_ints(chunk_a, currentChunkSize);
        random_ints(chunk_b, currentChunkSize);

        hipMemcpy(d_a, chunk_a, currentChunkSize * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_b, chunk_b, currentChunkSize * sizeof(int), hipMemcpyHostToDevice);

        // Launch the kernel
        vectorAdd << <numBlocks, BLOCK_SIZE >> > (d_a, d_b, d_c, currentChunkSize);

        // Copy result back
        hipMemcpy(chunk_c, d_c, currentChunkSize * sizeof(int), hipMemcpyDeviceToHost);

        // Optional: Process chunk_c here
    }

    printf("\nFirst elements of the result vector:\n");
    for (int i = 0; i < 100; i++) {
        printf("Element %d    %d + %d = %d\n", i, chunk_a[i], chunk_b[i], chunk_c[i]);
    }

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(chunk_a);
    free(chunk_b);
    free(chunk_c);

    return 0;
}
