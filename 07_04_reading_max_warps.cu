#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int device;
    hipGetDevice(&device); // Get current CUDA device
    hipDeviceProp_t prop;

    hipGetDeviceProperties(&prop, device);
    printf("Max_threads_per_SM  0: %d \n", prop.maxThreadsPerMultiProcessor);
    printf("Max_warps_per_SM    0: %d \n\n\n", (prop.maxThreadsPerMultiProcessor)/32);




    int maxThreadsPerMP = 0;
    hipDeviceGetAttribute(&maxThreadsPerMP, hipDeviceAttributeMaxThreadsPerMultiProcessor, device);
    printf("Max_threads_per_SM  1: %d   \n", maxThreadsPerMP);
    printf("Max_warps_per_SM    1: %d   \n", maxThreadsPerMP/32);

    return 0;
}