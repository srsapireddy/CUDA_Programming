#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void test01()
{
    //print the blocks and threads IDs
    // warp=32threads. (128 threads/block) --> ( 128/32 = 4 warps/block)
    int warp_ID_Value = 0;
    warp_ID_Value = threadIdx.x / 32;//%
    printf("The block ID is %d --- The thread ID is %d --- The warp ID %d\n",blockIdx.x,threadIdx.x, warp_ID_Value);
}

int main()
{   //add two vectors has 2048 elements
    // kernel_name<<<num_of_blocks , Num_of_threads_per_block>>>();
    //test01 <<<1, 2048>>> ();---- warps/block=2warps, total warps/GPU=4warps
    
    test01 <<<2, 64 >>> ();
    hipDeviceSynchronize();
    return 0;
}