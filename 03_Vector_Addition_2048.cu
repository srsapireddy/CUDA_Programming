#include <stdio.h>
#include <hip/hip_runtime.h>


#define SIZE 2048  // Define the size of the vectors

// CUDA Kernel for vector addition
__global__ void vectorAdd(int* A, int* B, int* C, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
        C[i] = A[i] + B[i];
}

int main() {
    int* A, * B, * C;            // Host vectors
    int* d_A, * d_B, * d_C;      // Device vectors
    int size = SIZE * sizeof(int);

    // Allocate and initialize host vectors
    A = (int*)malloc(size);
    B = (int*)malloc(size);
    C = (int*)malloc(size);



    // Allocate device vectors
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    for (int i = 0; i < SIZE; i++) {
        A[i] = i;
        B[i] = SIZE - i;
    }

    // Copy host vectors to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    vectorAdd <<<2, 1024 >>> (d_A, d_B, d_C, SIZE);

    // Copy result back to host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Calculate and print the execution time
    printf("\nExecution finished\n");
    for (int i = 0; i < SIZE; i++) {
        printf("%d + %d = %d  \n", A[i], B[i], C[i]);
    }

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);

    return 0;
}