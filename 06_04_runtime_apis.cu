#include <stdio.h>

int main() {
    int nDevices;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("  Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
            prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
            2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
        printf("  Total global memory: %lu\n", prop.totalGlobalMem);
        printf("  Compute capability: %d.%d\n",
            prop.major, prop.minor);
        printf("  Number of SMs: %d\n",
            prop.multiProcessorCount);
        printf("  Max threads per block: %d\n",
            prop.maxThreadsPerBlock);
        printf("  Max threads dimensions: x = %d, y = %d, z = %d\n",
            prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("  Max grid dimensions (Max number of Block per Grid): x = %d, y = %d, z = %d\n",
            prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    }
    return 0;
}